#include "hip/hip_runtime.h"
// ======================================================================== //
// Copyright 2018-2023 Ingo Wald                                            //
//                                                                          //
// Licensed under the Apache License, Version 2.0 (the "License");          //
// you may not use this file except in compliance with the License.         //
// You may obtain a copy of the License at                                  //
//                                                                          //
//     http://www.apache.org/licenses/LICENSE-2.0                           //
//                                                                          //
// Unless required by applicable law or agreed to in writing, software      //
// distributed under the License is distributed on an "AS IS" BASIS,        //
// WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied. //
// See the License for the specific language governing permissions and      //
// limitations under the License.                                           //
// ======================================================================== //

// helper stuff to more easily create test geometry
#include "owl/common/math/AffineSpace.h"
// primer itself
#include "primer/primer.h"
// std stuff
#include <vector>
#include <iostream>
#define STB_IMAGE_WRITE_IMPLEMENTATION
#include "stb/stb_image_write.h"
#include <random>

using namespace owl::common;

unsigned char to255(float f)
{
  if (f <= 0.f) return 0;
  if (f >= 1.f) return 255;
  return (unsigned char)(f*255.9f);
}

void savePNG(const std::string &fileName,
             vec2i fbSize,
             const vec3f *pixels)
{
  std::vector<unsigned char> rgba;
  for (int iy=fbSize.y-1;iy>=0;--iy) 
    for (int ix=0;ix<fbSize.x;ix++) {
      vec3f pixel = pixels[ix+fbSize.x*iy];
      rgba.push_back(to255(pixel.x));
      rgba.push_back(to255(pixel.y));
      rgba.push_back(to255(pixel.z));
      rgba.push_back(255);
    }
  std::cout << "#owl-prime.rtWeekend: writing image " << fileName << std::endl;
  stbi_write_png(fileName.c_str(),fbSize.x,fbSize.y,4,
                 rgba.data(),fbSize.x*sizeof(uint32_t));
}

inline float randomFloat()
{
  static std::random_device rd;  // Will be used to obtain a seed for the random number engine
  static std::mt19937 gen(rd()); // Standard mersenne_twister_engine seeded with rd()
  static std::uniform_real_distribution<> dis(0.f, 1.f);
  return dis(gen);
}

inline vec3f randomPoint()
{
  return vec3f(randomFloat(),randomFloat(),randomFloat());
}

inline vec3f randomDirection()
{
  while (true) {
    vec3f v = 2.f*randomPoint()-vec3f(1.f);
    if (dot(v,v) <= 1.f)
      return normalize(v);
  }
}

int main(int, char **)
{
  std::cout << "creating primer context" << std::endl;
  OPContext context
    = opContextCreate(OP_CONTEXT_DEFAULT,0);

  std::vector<float4> spheres;
  for (int i=0;i<100;i++) {
    vec3f P = 8.f*(randomPoint()-vec3f(.2f));
    float r = .2f + sqrtf(.3f*randomFloat());
    spheres.push_back(make_float4(P.x,P.y,P.z,r));
  }

  
  OPGeom geom
    = opSpheres4f(context,
                  /* user-supplied geometry ID */ 0,
                  spheres.data(),spheres.size());

  OPModel model
    = opModelFromGeoms(context,
                       &geom,1);
  
  std::cout << "generating rays..." << std::endl;
  vec3f up(0,1,0);
  vec3f at(0,0,0);
  vec3f from(-3,-2,-1);

  vec2i fbSize(800,600);
  vec3f dir = normalize(at-from);
  float imagePlaneHeight = 10.f;
  vec3f horiz = (imagePlaneHeight * fbSize.x/float(fbSize.y)) * normalize(cross(dir,up));
  vec3f vert  = imagePlaneHeight * normalize(cross(horiz,dir));

  std::vector<OPRay> rays;
  for (int iy=0;iy<fbSize.y;iy++)
    for (int ix=0;ix<fbSize.x;ix++) {
      float du = (ix+.5f)/fbSize.x;
      float dv = (iy+.5f)/fbSize.y;
      OPRay ray;
      (vec3f&)ray.origin = from + (du-.5f) * horiz + (dv-.5f) * vert;
      (vec3f&)ray.direction = dir;
      ray.tMin = 0.f;
      ray.tMax = INFINITY;
      rays.push_back(ray);
    }
  std::cout << "tracing rays..." << std::endl;
  std::vector<OPHit> hits(rays.size());
  opTrace(model,rays.size(),rays.data(),hits.data(),OP_TRACE_FLAGS_DEFAULT);

  std::vector<vec3f> pixels(rays.size());
  // ------------------------------------------------------------------
  std::cout << "creating 'primID' image" << std::endl;
  for (int i=0;i<rays.size();i++)
    pixels[i] = randomColor(hits[i].primID);
  savePNG("opSampleSpheres_primIDs.png",fbSize,pixels.data());
}
