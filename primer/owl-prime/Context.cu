#include "hip/hip_runtime.h"
// ======================================================================== //
// Copyright 2019-2023 Ingo Wald                                            //
//                                                                          //
// Licensed under the Apache License, Version 2.0 (the "License");          //
// you may not use this file except in compliance with the License.         //
// You may obtain a copy of the License at                                  //
//                                                                          //
//     http://www.apache.org/licenses/LICENSE-2.0                           //
//                                                                          //
// Unless required by applicable law or agreed to in writing, software      //
// distributed under the License is distributed on an "AS IS" BASIS,        //
// WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied. //
// See the License for the specific language governing permissions and      //
// limitations under the License.                                           //
// ======================================================================== //

#include "owl-prime/Context.h"
#include "owl-prime/Triangles.h"
#include "owl-prime/Spheres.h"
#include "owl-prime/Group.h"

primer::Context *primer::Context::createOffloadContext(int gpuID)
{ return new op::Context(gpuID); }

extern "C" char deviceCode_ptx[];

namespace op {
  
  OWLVarDecl Context::lpVariables[]
  = {
     { "rays",      OWL_RAW_POINTER, OWL_OFFSETOF(Context::LPData,rays) },
     { "hits",      OWL_RAW_POINTER, OWL_OFFSETOF(Context::LPData,hits) },
     { "activeIDs", OWL_RAW_POINTER, OWL_OFFSETOF(Context::LPData,activeIDs) },
     { "model",     OWL_GROUP,       OWL_OFFSETOF(Context::LPData,model) },
     { "numRays",   OWL_INT,         OWL_OFFSETOF(Context::LPData,numRays) },
     { "isSOA",     OWL_INT,         OWL_OFFSETOF(Context::LPData,isSOA) },
     { "numHitsPerRay",   OWL_INT,   OWL_OFFSETOF(Context::LPData,numHitsPerRay) },
     { "flags",     OWL_ULONG,       OWL_OFFSETOF(Context::LPData,flags) },
     { nullptr /* end of list sentinel */ }
  };

  Context::Context(int gpuID)
  {
    if (gpuID < 0) gpuID = 0;
    
    owl = owlContextCreate(&gpuID,1);
    module = owlModuleCreate(owl,deviceCode_ptx);
    rayGen = owlRayGenCreate(owl,module,"traceRays",sizeof(int),nullptr,0);
    launchParams = owlParamsCreate(owl,sizeof(LPData),lpVariables,-1);
    
    meshGeomType = owlGeomTypeCreate(owl,OWL_TRIANGLES,
                                     sizeof(Triangles::SBTData),
                                     Triangles::variables,-1);
    owlGeomTypeSetClosestHit(meshGeomType,0,module,"Triangles");
    owlGeomTypeSetAnyHit(meshGeomType,0,module,"MultiHit");

    spheresGeomType = owlGeomTypeCreate(owl,OWL_GEOM_USER,
                                        sizeof(Spheres::SBTData),
                                        Spheres::variables,-1);
    owlGeomTypeSetBoundsProg(spheresGeomType,module,"Spheres");
    owlGeomTypeSetIntersectProg(spheresGeomType,0,module,"Spheres");
    owlGeomTypeSetClosestHit(spheresGeomType,0,module,"Spheres");
    owlGeomTypeSetAnyHit(spheresGeomType,0,module,"MultiHit");

    owlBuildPrograms(owl);
    owlBuildPipeline(owl);
  }
  
  void Context::checkSBT()
  {
    if (!sbtDirty) return;

    owlBuildSBT(owl);
    owlBuildPipeline(owl);
    sbtDirty = false;
  }

  template<typename T>
  inline __both__
  const T &getWithOffset(const T *base, int idx, size_t strideInBytes)
  {
    unsigned char *ptr = (unsigned char *)base;
    ptr += idx * strideInBytes;
    return *(T*)ptr;
  }

  __global__ void copySpheres(float4 *spheres,
                              int numSpheres,
                              const float *x,
                              const float *y,
                              const float *z,
                              int centerStride,
                              const float *r,
                              int rStride)
  {
    int tid = threadIdx.x+blockIdx.x*blockDim.x;
    if (tid >= numSpheres) return;
    spheres[tid]
      = {
         getWithOffset(x,tid,centerStride),
         getWithOffset(y,tid,centerStride),
         getWithOffset(z,tid,centerStride),
         getWithOffset(r,tid,rStride)
    };
  }

  /*! creates a new spheres geometry from the given centers and radii */
  primer::Geom *Context::createSpheres(uint32_t userID,
                                       int numSpheres,
                                       /* vertex array */
                                       const float *xArray, 
                                       const float *yArray, 
                                       const float *zArray,
                                       size_t strideCenters,
                                       const float *rArray,
                                       size_t strideRadii)
  {
    OWLBuffer spheresBuffer = 0;
    if (isDeviceAccessible(xArray)) {
      spheresBuffer = owlDeviceBufferCreate(owl,OWL_FLOAT4,numSpheres,0);
      copySpheres<<<divRoundUp(numSpheres,1024),1024>>>
        ((float4*)owlBufferGetPointer(spheresBuffer,0),
         numSpheres,
         xArray,
         yArray,
         zArray,strideCenters,
         rArray,strideRadii);
    } else {
      std::vector<float4> aos(numSpheres);
      for (int i=0;i<numSpheres;i++) {
        aos[i] = {
                  getWithOffset(xArray,i,strideCenters),
                  getWithOffset(yArray,i,strideCenters),
                  getWithOffset(zArray,i,strideCenters),
                  getWithOffset(rArray,i,strideRadii)
        };
      }
      spheresBuffer = owlDeviceBufferCreate(owl,OWL_FLOAT4,numSpheres,aos.data());
    }
    return new Spheres(this,userID,
                       spheresBuffer,numSpheres);
    
  }
  
  primer::Geom *Context::createTriangles(uint32_t userID,
                                         size_t numTriangles,
                                         const float *v0x, 
                                         const float *v0y, 
                                         const float *v0z, 
                                         const float *v1x, 
                                         const float *v1y, 
                                         const float *v1z, 
                                         const float *v2x, 
                                         const float *v2y, 
                                         const float *v2z,
                                         size_t strideInBytes)
  {
    OWLBuffer vertexBuffer
      = owlManagedMemoryBufferCreate(owl,OWL_FLOAT3,3*numTriangles,0);
    vec3f *vertices = (vec3f*)owlBufferGetPointer(vertexBuffer,0);
    for (int i=0;i<numTriangles;i++) {
      vertices[3*i+0] = {
                         getWithOffset(v0x,i,strideInBytes),
                         getWithOffset(v0y,i,strideInBytes),
                         getWithOffset(v0z,i,strideInBytes)
      };
      vertices[3*i+1] = {
                         getWithOffset(v1x,i,strideInBytes),
                         getWithOffset(v1y,i,strideInBytes),
                         getWithOffset(v1z,i,strideInBytes)
      };
      vertices[3*i+2] = {
                         getWithOffset(v2x,i,strideInBytes),
                         getWithOffset(v2y,i,strideInBytes),
                         getWithOffset(v2z,i,strideInBytes)
      };
    }
    
    OWLBuffer indexBuffer
      = owlManagedMemoryBufferCreate(owl,OWL_INT3,numTriangles,0);
    
    vec3i *indices = (vec3i*)owlBufferGetPointer(indexBuffer,0);
    // TODO: do that in CUDA (so would also work on device pointers)
    for (int i=0;i<numTriangles;i++) {
      indices[i] = vec3i(3*i)+vec3i{0,1,2};
    }

    return new Triangles(this,userID,
                         vertexBuffer,3*numTriangles,
                         indexBuffer,numTriangles);
  }

  /*! create a mesh from vertex array and index array */
  primer::Geom *Context::createTriangles(uint32_t userID,
                                         /* vertex array */
                                         const vec3f *vertices,
                                         size_t numVertices,
                                         size_t vertexStrideInBytes,
                                         /* index array */
                                         const vec3i *indices,
                                         size_t numIndices,
                                         size_t indexStrideInBytes)
  {
    // TODO: do all this without copies if these are already device pointers
    OWLBuffer vertexBuffer
      = owlManagedMemoryBufferCreate(owl,OWL_FLOAT3,numVertices,0);
    vec3f *d_vertices = (vec3f*)owlBufferGetPointer(vertexBuffer,0);
    
    OWLBuffer indexBuffer
      = owlManagedMemoryBufferCreate(owl,OWL_INT3,numIndices,0);
    vec3i *d_indices = (vec3i*)owlBufferGetPointer(indexBuffer,0);

    for (int i=0;i<numVertices;i++) 
      d_vertices[i] = getWithOffset(vertices,i,vertexStrideInBytes);
    for (int i=0;i<numIndices;i++) 
      d_indices[i] = getWithOffset(indices,i,indexStrideInBytes);
    
    return new Triangles(this,userID,
                         vertexBuffer,numVertices,
                         indexBuffer,numIndices);
  }

  primer::Group *Context::createGroup(std::vector<OPGeom> &geoms) 
  {
    return new op::Group(this,geoms);
  }

  primer::Model *Context::createModel(const std::vector<OPGroup>  &groups,
                                      const std::vector<affine3f> &xfms,
                                      const std::vector<int>      &userIDs)
  {
    return new op::Model(this,groups,xfms,userIDs);
  }
  
} // ::op
