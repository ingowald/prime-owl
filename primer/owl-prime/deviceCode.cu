#include "hip/hip_runtime.h"
// ======================================================================== //
// Copyright 2019-2022 Ingo Wald                                            //
//                                                                          //
// Licensed under the Apache License, Version 2.0 (the "License");          //
// you may not use this file except in compliance with the License.         //
// You may obtain a copy of the License at                                  //
//                                                                          //
//     http://www.apache.org/licenses/LICENSE-2.0                           //
//                                                                          //
// Unless required by applicable law or agreed to in writing, software      //
// distributed under the License is distributed on an "AS IS" BASIS,        //
// WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied. //
// See the License for the specific language governing permissions and      //
// limitations under the License.                                           //
// ======================================================================== //

#include "owl-prime/deviceCode.h"
#include "owl-prime/Context.h"
#include "owl-prime/Triangles.h"

using namespace owl::common;
using op::Hit;

extern "C" __constant__ op::Context::LPData optixLaunchParams;

struct MultiHitPRD
{
  /*! the list of hits to be used for this trace */
  Hit  *hits;
  /*! number of hits already stored in the list */
  int   numHitsFound;
  /*! the (current) maximum t after which we *know* no more closer
      hits can be found. this is either the last entry in the list (if
      it is full), or ray.tmax (if not) */
  float cutOff;
};

  
OPTIX_BOUNDS_PROGRAM(Spheres)(const void  *geomData,
                              box3f       &primBounds,
                              const int    primID)
{
  const auto &self = *(const op::Spheres::SBTData*)geomData;
  float4 sphere = self.spheres[primID];
  primBounds.lower.x = sphere.x - sphere.w;
  primBounds.lower.y = sphere.y - sphere.w;
  primBounds.lower.z = sphere.z - sphere.w;
  primBounds.upper.x = sphere.x + sphere.w;
  primBounds.upper.y = sphere.y + sphere.w;
  primBounds.upper.z = sphere.z + sphere.w;
}

OPTIX_INTERSECT_PROGRAM(Spheres)()
{
  const auto &self
    = owl::getProgramData<op::Spheres::SBTData>();
  const int primID = optixGetPrimitiveIndex();
  
  const vec3f org  = optixGetWorldRayOrigin();
  const vec3f dir  = optixGetWorldRayDirection();
  float hit_t      = optixGetRayTmax();
  const float tmin = optixGetRayTmin();

  const float4 sphere = self.spheres[primID];
  const vec3f center = {sphere.x,sphere.y,sphere.z};
  const float radius = sphere.w;
  const vec3f oc = org - center;
         
  const float a = dot(dir,dir);
  const float b = dot(oc, dir);
  const float c = dot(oc, oc) - radius * radius;
  const float discriminant = b * b - a * c;
  
  if (discriminant < 0.f) return;

  {
    float temp = (-b - sqrtf(discriminant)) / a;
    if (temp < hit_t && temp > tmin) 
      hit_t = temp;
  }
      
  {
    float temp = (-b + sqrtf(discriminant)) / a;
    if (temp < hit_t && temp > tmin) 
      hit_t = temp;
  }
  if (hit_t < optixGetRayTmax()) {
    optixReportIntersection(hit_t, 0);
  }
}

/*! closest hit program: fills in a hit structure for the current ray */
OPTIX_CLOSEST_HIT_PROGRAM(Spheres)()
{
  const auto &self = owl::getProgramData<op::Spheres::SBTData>();
  Hit &hit = owl::getPRD<Hit>();
  hit.t = optixGetRayTmax();
  hit.primID = optixGetPrimitiveIndex();
  hit.geomID = self.userID;
  hit.instID = optixGetInstanceIndex();
  hit.u      = 0.f;
  hit.v      = 0.f;
}

/*! closest hit program: fills in a hit structure for the current ray */
OPTIX_CLOSEST_HIT_PROGRAM(Triangles)()
{
  const auto &self  = owl::getProgramData<op::Triangles::SBTData>();
  Hit &hit = owl::getPRD<Hit>();
  hit.t = optixGetRayTmax();
  hit.primID = optixGetPrimitiveIndex();
  hit.geomID = self.userID;
  hit.instID = optixGetInstanceIndex();
  hit.u      = optixGetTriangleBarycentrics().x;
  hit.v      = optixGetTriangleBarycentrics().y;
}

/*! closest hit program: used for the multi-hit kernel with up to N
    hits per ray. this should never get called if not in a multi-hit
    trace (we use the disable_ah flag to turn it off for other
    queries), so inside this code we can safely assume that there's a
    multi-hit PRD */
OPTIX_ANY_HIT_PROGRAM(MultiHit)()
{
  const auto &self = owl::getProgramData<op::Triangles::SBTData>();
  // if (lp.numHitsPerRay == 0) {
  //   // NOT multi-hit, but TRACE_CONTINUE
  //   Hit &hit = owl::getPRD<Hit>();
  // } else {
    MultiHitPRD &prd = owl::getPRD<MultiHitPRD>();
    Hit *hitList = prd.hits;

    Hit thisHit;
    thisHit.t      = optixGetRayTmax();
    thisHit.primID = optixGetPrimitiveIndex();
    thisHit.geomID = self.userID;
    thisHit.instID = optixGetInstanceIndex();
    thisHit.u      = optixGetTriangleBarycentrics().x;
    thisHit.v      = optixGetTriangleBarycentrics().y;
    if (thisHit.t >= prd.cutOff) {
      // ACCEPT hit to optix (this will move new ray.tmax to thisHit.t),
      // but don't save in list.
      return;
    } else {
      // we KNOW that the list is not yet full, OR that this hit is
      // closer than the furthest so-far found one.
      auto &lp = optixLaunchParams;
      int insertPos
        = (prd.numHitsFound < lp.numHitsPerRay)
        ? /* list not yet full: append*/prd.numHitsFound
        : /* list full, overwrite furthest*/(lp.numHitsPerRay-1);
      while (insertPos > 0 && hitList[insertPos-1].t > thisHit.t) {
        hitList[insertPos] = hitList[insertPos-1];
        --insertPos;
      }
      hitList[insertPos] = thisHit;
      prd.numHitsFound++;
      if (prd.numHitsFound >= lp.numHitsPerRay) {
        prd.cutOff = min(prd.cutOff,hitList[lp.numHitsPerRay-1].t);
        prd.numHitsFound = lp.numHitsPerRay;
      }
      if (thisHit.t < prd.cutOff)
        optixIgnoreIntersection();
    }
  // }
}

OPTIX_RAYGEN_PROGRAM(traceRays)()
{
  Hit hit;
  hit.clear();
  int rayID
    = owl::getLaunchIndex().x
    + owl::getLaunchDims().x
    * owl::getLaunchIndex().y;

  auto &lp = optixLaunchParams;
  if (rayID >= lp.numRays) return;

  if (lp.activeIDs) {
    rayID = lp.activeIDs[rayID];
    if (rayID < 0) return;
  }
  
  owl::Ray ray;
  if (lp.isSOA) {
    ray = owl::Ray(vec3f(lp.soa.ray.org_x[rayID],
                         lp.soa.ray.org_y[rayID],
                         lp.soa.ray.org_z[rayID]),
                   vec3f(lp.soa.ray.dir_x[rayID],
                         lp.soa.ray.dir_y[rayID],
                         lp.soa.ray.dir_z[rayID]),
                   lp.soa.ray.t_min[rayID],
                   lp.soa.ray.t_max[rayID]);
  } else {
    ray = owl::Ray((const vec3f&)lp.rays[rayID].origin,
                   (const vec3f&)lp.rays[rayID].direction,
                   lp.rays[rayID].tMin,
                   lp.rays[rayID].tMax);
  }
  if (ray.tmin < ray.tmax) {
    if (lp.numHitsPerRay == 0) {
      if (lp.flags & OP_TRACE_CONTINUE) {
        uint32_t rayFlags = 0;
        owl::traceRay(lp.model,ray,hit,rayFlags);
      } else {
        // this is the "defualt" path where each ray finds _the_ closest hit
        uint32_t rayFlags = OPTIX_RAY_FLAG_DISABLE_ANYHIT;
        if (lp.flags & OP_TRACE_FLAGS_TERMINATE_ON_FIRST_HIT)
          rayFlags |= OPTIX_RAY_FLAG_TERMINATE_ON_FIRST_HIT;
        owl::traceRay(lp.model,ray,hit,rayFlags);
      }
    } else {
      // in this path each ray can find and store up to N different
      // hits; we do this via a anyhit program (and disable closest hit)
      uint32_t rayFlags = OPTIX_RAY_FLAG_DISABLE_CLOSESTHIT;
      MultiHitPRD multiHitPrd;
      multiHitPrd.numHitsFound = 0;
      multiHitPrd.cutOff       = ray.tmax;
      multiHitPrd.hits         = lp.hits+lp.numHitsPerRay*rayID;
      owl::traceRay(lp.model,ray,multiHitPrd,rayFlags);
      for (int i=multiHitPrd.numHitsFound; i<lp.numHitsPerRay; i++)
        multiHitPrd.hits[i].primID = -1;
    }
  }
  if (lp.isSOA) {
    if (lp.soa.hit.primID)
      lp.soa.hit.primID[rayID] = hit.primID;
    if (lp.soa.hit.geomID)
      lp.soa.hit.geomID[rayID] = hit.geomID;
    if (lp.soa.hit.instID)
      lp.soa.hit.instID[rayID] = hit.instID;
  } else {
    lp.hits[rayID] = hit;
  }
}


